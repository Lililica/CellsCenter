#include "hip/hip_runtime.h"
#include "LlyodCentralisation.hpp"
#include <iostream>	
#include <hip/hip_runtime.h>
#include <vector>


__global__ void centralePointLLoyd(float* pointList, int* adjacencySize, int* pointAdjacencyList)
{
    int idx = threadIdx.x * 2;
    int idxAdjacencyStart = adjacencySize[idx];
    int nbrOfAdjacencies = adjacencySize[idx + 1];

    if(nbrOfAdjacencies <= 0) {
        // If there are no adjacencies, we can skip the computation
        return;
    }

    float moyenneX = 0.0f;
    float moyenneY = 0.0f;

    for (int i = 0; i < nbrOfAdjacencies; ++i) {
        int currentAdjacencyIndex = 2*pointAdjacencyList[idxAdjacencyStart + i];
        moyenneX += pointList[currentAdjacencyIndex];
        moyenneY += pointList[currentAdjacencyIndex + 1];
    }

    if (nbrOfAdjacencies > 0) {
        moyenneX /= nbrOfAdjacencies;
        moyenneY /= nbrOfAdjacencies;
    }

    pointList[idx] = moyenneX;
    pointList[idx + 1] = moyenneY;
    
}

template<typename T>
__host__ void print_matrix(T* matrix, int size)
{
    for (int i = 0; i < size; ++i) {
        std::cout << matrix[i] << " ";
    }
    std::cout << std::endl;
}

template<typename T>
T* vectorToCArray(const std::vector<T>& vec) {
    T* arr = new T[vec.size()];
    std::copy(vec.begin(), vec.end(), arr);
    return arr;
}

template<typename T>
std::vector<T> cArrayToVector(const T* arr, size_t size) {
    return std::vector<T>(arr, arr + size);
}

void Graphe::centralisation()
{
    std::cout << "Starting centralisation : ------------------------------------------------------ " << std::endl;

    if(pointList.size() == 0 || adjacencySize.size() == 0 || pointAdjacencyList.size() == 0) {
        std::cerr << "Error: One of the input vectors is empty." << std::endl;
        return;
    }

    if(pointList.size() !=  adjacencySize.size()) {
        std::cerr << "Error: Wrong vector." << std::endl;
        return;
    }


    float* pointListCARRAY = vectorToCArray<float>(pointList);
    int* adjacencySizeCARRAY = vectorToCArray<int>(adjacencySize);
    int* pointAdjacencyListCARRAY = vectorToCArray<int>(pointAdjacencyList);


    std::cout<<"pointListCARRAY =\n";
    print_matrix<float>(pointListCARRAY, pointList.size());
    std::cout<<"adjacencySizeCARRAY =\n";
    print_matrix<int>(adjacencySizeCARRAY , adjacencySize.size());
    std::cout<<"pointAdjacencyListCARRAY =\n";
    print_matrix<int>(pointAdjacencyListCARRAY  , pointAdjacencyList.size());

    /* Setting up variables on device. i.e. GPU */
    float* pointListCUDA;
    int *adjacencySizeCUDA, *pointAdjacencyListCUDA;
    hipMalloc((void**)&pointListCUDA, pointList.size() * sizeof(float));
    hipMalloc((void**)&adjacencySizeCUDA, adjacencySize.size() * sizeof(int));
    hipMalloc((void**)&pointAdjacencyListCUDA, pointAdjacencyList.size() * sizeof(int));

    /* Copy data from host to device */
    hipMemcpy(pointListCUDA, pointListCARRAY, pointList.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(adjacencySizeCUDA, adjacencySizeCARRAY, adjacencySize.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pointAdjacencyListCUDA, pointAdjacencyListCARRAY, pointAdjacencyList.size() * sizeof(int), hipMemcpyHostToDevice);
    
    /* 
    Kernel Launch
    Grid contains M*N blocks
    Each block has 1 thread
    Hence index of matrix element is
    blockIdx.x* gridSize.y + blockIdx.y
    */
    int nbrThreads = pointList.size()/2; // Assuming each block has 2 threads
    dim3 gridSize((pointList.size() + nbrThreads - 1)/nbrThreads, 1); // Assuming each block has 64 threads
    centralePointLLoyd<<<gridSize, nbrThreads>>>(pointListCUDA, adjacencySizeCUDA, pointAdjacencyListCUDA);
    hipDeviceSynchronize();

    /* Copy result from GPU device to host */
    float* pointListResult = new float[pointList.size()];    
    hipMemcpy(pointListResult, pointListCUDA, pointList.size() * sizeof(float), hipMemcpyDeviceToHost);
    
    /* Print result */
    std::cout << "Resulting pointList after centralisation:\n";
    print_matrix<float>(pointListResult, pointList.size());

    /* Convert result back to vector */
    pointList = cArrayToVector<float>(pointListResult, pointList.size());

    /* Cleanup device and host memory */
    hipFree(pointListCUDA);
    hipFree(adjacencySizeCUDA);
    hipFree(pointAdjacencyListCUDA);
    delete[] pointListCARRAY;
    delete[] adjacencySizeCARRAY;
    delete[] pointAdjacencyListCARRAY;
    delete[] pointListResult;

    std::cout << "End of centralisation : ------------------------------------------------------ " << std::endl;

}