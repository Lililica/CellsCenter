#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include "LlyodCentralisation.hpp"

__global__ void centralePointLLoyd(float* pointList, int* adjacencySize, int* pointAdjacencyList)
{
    int idx               = threadIdx.x * 2;
    int idxAdjacencyStart = adjacencySize[idx];
    int nbrOfAdjacencies  = adjacencySize[idx + 1];

    if (nbrOfAdjacencies <= 0)
    {
        // If there are no adjacencies, we can skip the computation
        return;
    }

    float moyenneX = 0.0f;
    float moyenneY = 0.0f;

    for (int i = 0; i < nbrOfAdjacencies; ++i)
    {
        int currentAdjacencyIndex = 2 * pointAdjacencyList[idxAdjacencyStart + i];
        moyenneX += pointList[currentAdjacencyIndex];
        moyenneY += pointList[currentAdjacencyIndex + 1];
    }

    if (nbrOfAdjacencies > 0)
    {
        moyenneX /= nbrOfAdjacencies;
        moyenneY /= nbrOfAdjacencies;
    }

    pointList[idx]     = moyenneX;
    pointList[idx + 1] = moyenneY;
}

template<typename T>
__host__ void print_matrix(T* matrix, int size)
{
    for (int i = 0; i < size; ++i)
    {
        std::cout << matrix[i] << " ";
    }
    std::cout << std::endl;
}

template<typename T>
T* vectorToCArray(const std::vector<T>& vec)
{
    T* arr = new T[vec.size()];
    std::copy(vec.begin(), vec.end(), arr);
    return arr;
}

template<typename T>
std::vector<T> cArrayToVector(const T* arr, size_t size)
{
    return std::vector<T>(arr, arr + size);
}

template<typename T, size_t N>
T* arrayToCArray(const std::array<T, N>& arr)
{
    T* cArr = new T[N];
    std::copy(arr.begin(), arr.end(), cArr);
    return cArr;
}

template<typename T, size_t N>
std::array<T, N> cArrayToArray(const T* cArr)
{
    std::array<T, N> arr;
    std::copy(cArr, cArr + N, arr.begin());
    return arr;
}

void Graphe::centralisation()
{
    std::cout << "Starting centralisation : ------------------------------------------------------ " << std::endl;

    if (pointList.size() == 0 || adjacencySize.size() == 0 || pointAdjacencyList.size() == 0)
    {
        std::cerr << "Error: One of the input vectors is empty." << std::endl;
        return;
    }

    if (pointList.size() != adjacencySize.size())
    {
        std::cerr << "Error: Wrong vector." << std::endl;
        std::cerr << "pointList.size() = " << pointList.size() << ", adjacencySize.size() = " << adjacencySize.size() << std::endl;
        return;
    }

    float* pointListCARRAY          = vectorToCArray<float>(pointList);
    int*   adjacencySizeCARRAY      = vectorToCArray<int>(adjacencySize);
    int*   pointAdjacencyListCARRAY = vectorToCArray<int>(pointAdjacencyList);

    // std::cout << "pointListCARRAY =\n";
    // print_matrix<float>(pointListCARRAY, pointList.size());
    // std::cout << "adjacencySizeCARRAY =\n";
    // print_matrix<int>(adjacencySizeCARRAY, adjacencySize.size());
    // std::cout << "pointAdjacencyListCARRAY =\n";
    // print_matrix<int>(pointAdjacencyListCARRAY, pointAdjacencyList.size());

    /* Setting up variables on device. i.e. GPU */
    float* pointListCUDA;
    int *  adjacencySizeCUDA, *pointAdjacencyListCUDA;
    hipMalloc((void**)&pointListCUDA, pointList.size() * sizeof(float));
    hipMalloc((void**)&adjacencySizeCUDA, adjacencySize.size() * sizeof(int));
    hipMalloc((void**)&pointAdjacencyListCUDA, pointAdjacencyList.size() * sizeof(int));

    /* Copy data from host to device */
    hipMemcpy(pointListCUDA, pointListCARRAY, pointList.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(adjacencySizeCUDA, adjacencySizeCARRAY, adjacencySize.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pointAdjacencyListCUDA, pointAdjacencyListCARRAY, pointAdjacencyList.size() * sizeof(int), hipMemcpyHostToDevice);

    /*
    Kernel Launch
    Grid contains M*N blocks
    Each block has 1 thread
    Hence index of matrix element is
    blockIdx.x* gridSize.y + blockIdx.y
    */
    int  nbrThreads = pointList.size() / 2;                             // Assuming each block has 2 threads
    dim3 gridSize((pointList.size() + nbrThreads - 1) / nbrThreads, 1); // Assuming each block has 64 threads
    centralePointLLoyd<<<gridSize, nbrThreads>>>(pointListCUDA, adjacencySizeCUDA, pointAdjacencyListCUDA);
    hipDeviceSynchronize();

    /* Copy result from GPU device to host */
    float* pointListResult = new float[pointList.size()];
    hipMemcpy(pointListResult, pointListCUDA, pointList.size() * sizeof(float), hipMemcpyDeviceToHost);

    /* Print result */
    // std::cout << "Resulting pointList after centralisation:\n";
    // print_matrix<float>(pointListResult, pointList.size());

    /* Convert result back to vector */
    pointList = cArrayToVector<float>(pointListResult, pointList.size());

    /* Cleanup device and host memory */
    hipFree(pointListCUDA);
    hipFree(adjacencySizeCUDA);
    hipFree(pointAdjacencyListCUDA);
    delete[] pointListCARRAY;
    delete[] adjacencySizeCARRAY;
    delete[] pointAdjacencyListCARRAY;
    delete[] pointListResult;

    std::cout << "End of centralisation : ------------------------------------------------------ " << std::endl;
}

void Graphe::init_from_bad_format(const std::vector<Point>& points, const std::vector<Adjacency>& adjacencies)
{
    std::cout << "Initializing from bad format: ------------------------------------------------------ " << std::endl;

    pointList.clear();
    adjacencySize.clear();
    pointAdjacencyList.clear();

    for (const Point& point : points)
    {
        pointList.emplace_back(point.first);
        pointList.emplace_back(point.second);

    }

    std::vector<std::pair<int,int>> adjacenciesIdx(adjacencies.size());

    for (int k = 0; k < adjacencies.size(); k++)
    {
        Adjacency adjacency = adjacencies[k];

        for(int i = 0; i < pointList.size(); i += 2)
        {
            Point currentPoint = std::pair<float,float>(pointList[i], pointList[i+1]);


            if(pointEqual(currentPoint, adjacency.first))
            {
                adjacenciesIdx[k].first = i/2;
            }
        
        
            if(pointEqual(currentPoint, adjacency.second))
            {
                adjacenciesIdx[k].second = i/2;
            }
        }
    }

    std::vector<std::vector<int>> AdjacencyList(points.size());


    for (int k = 0; k < adjacenciesIdx.size(); k++)
    {

        if(std::find(AdjacencyList[adjacenciesIdx[k].first].begin(), AdjacencyList[adjacenciesIdx[k].first].end(), adjacenciesIdx[k].second) == AdjacencyList[adjacenciesIdx[k].first].end())
        {
            AdjacencyList[adjacenciesIdx[k].first].emplace_back(adjacenciesIdx[k].second);
        }
        if(std::find(AdjacencyList[adjacenciesIdx[k].second].begin(), AdjacencyList[adjacenciesIdx[k].second].end(), adjacenciesIdx[k].first) == AdjacencyList[adjacenciesIdx[k].second].end())
        {
            AdjacencyList[adjacenciesIdx[k].second].emplace_back(adjacenciesIdx[k].first);
        }
    }

    for (int k = 0; k < AdjacencyList.size(); k++)
    {
        std::vector<int> adjacency = AdjacencyList[k];

        if(k == 0)
            adjacencySize.emplace_back(0);

        adjacencySize.emplace_back(int(adjacency.size()));

        if(k != AdjacencyList.size() - 1)
            adjacencySize.emplace_back(adjacencySize.back() + adjacencySize[adjacencySize.size() - 1]);
        
        for (int idx : adjacency)
        {
            pointAdjacencyList.emplace_back(idx);
        }
    }

    std::cout << "Initialization complete." << std::endl;
}
